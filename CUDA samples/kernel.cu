#include "hip/hip_runtime.h"
﻿/*
The goal of this task is to implement a CUDA program that computes a known integral (f(x) = 4/(1+x^2)) using the trapezoidal method.
*/
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void trapezoidalIntegration(double* results, double step, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        double x1 = tid * step;
        double x2 = (tid + 1) * step;
        results[tid] = (4.0 / (1.0 + x1 * x1) + 4.0 / (1.0 + x2 * x2)) * step / 2.0;    // f(x) = 4/(1+x^2)
    }
}

int main() {
    const int N = 10000000;
    const int blockSize = 256;
    const int gridSize = (N + blockSize - 1) / blockSize;

    double step = 1.0 / N;
    double* results;
    hipMallocManaged(&results, N * sizeof(double));

    trapezoidalIntegration << <gridSize, blockSize >> > (results, step, N);
    hipDeviceSynchronize();

    double sum = 0.0;
    for (int i = 0; i < N; ++i) {
        sum += results[i];
    }

    std::cout << "Wartość całki: " << sum << std::endl;

    hipFree(results);
    return 0;
}
