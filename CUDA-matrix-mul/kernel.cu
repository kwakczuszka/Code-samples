﻿/*
This program utilizes GPU to multiply two randomly generated square matrices (1024x1024) using standard O(N^2) algorithm.
*/
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <fstream>
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void matrixMul(const int* a, const int* b, int* c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void printMatrix(std::vector<int> matrix, int rows, int cols, std::ostream& os) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            os << matrix[i * rows + j] << " ";
        }
        os << "\n";
    }
}

void verify_result(std::vector<int>& a, std::vector<int>& b, std::vector<int>& c, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            assert(tmp == c[i * N + j]);    //check control sum
        }
    }
}

int main() {
    int N = 1 << 10;    // Matrix size of 1024 x 1024;

    size_t bytes = N * N * sizeof(int);

    std::vector<int> host_a(N * N);
    std::vector<int> host_b(N * N);
    std::vector<int> host_c(N * N);

    std::generate(host_a.begin(), host_a.end(), []() { return rand() % 100; });
    std::generate(host_b.begin(), host_b.end(), []() { return rand() % 100; });

    int* device_a, * device_b, * device_c;
    hipMalloc(&device_a, bytes);
    hipMalloc(&device_b, bytes);
    hipMalloc(&device_c, bytes);

    hipMemcpy(device_a, host_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;

    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);
    std::cout << "GPU calculation started\n";

    matrixMul <<<blocks, threads >>> (device_a, device_b, device_c, N);

    std::cout << "GPU calculation finished\n";
    hipMemcpy(host_c.data(), device_c, bytes, hipMemcpyDeviceToHost);
    std::cout << "CPU calculation started\n";

    verify_result(host_a, host_b, host_c, N);
    std::cout << "CPU calculation finished\n";

    std::ofstream fileout1("mat1.txt");
    std::ofstream fileout2("mat2.txt");
    std::ofstream fileout3("res.txt");

    printMatrix(host_a, N, N, fileout1);
    printMatrix(host_b, N, N, fileout2);
    printMatrix(host_c, N, N, fileout3);

    fileout1.close();
    fileout2.close();
    fileout3.close();

    std::cout << "COMPLETED SUCCESSFULLY\n";

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}