﻿
#include "hip/hip_runtime.h"


#include <vector>
#include <numeric>
#include <stdio.h>
#include <random>
#include <iostream>

hipError_t addWithCuda(int *a, int *b, int *out, unsigned int size);

__global__ void addKernel(int *a, int *b, int *out, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is not out of bounds
    if (idx < size) {
        out[idx] = a[idx] + b[idx];
    }
}

int main()
{
    const long arraySize = 10000000;
    int* a = new int[arraySize];
    int* b = new int[arraySize];
    int* out = new int[arraySize];

    std::random_device rd;
    std::mt19937 gen(rd());

    std::uniform_int_distribution<> dist(-100, 100);

    int control_sum1 = 0;
    for (int i = 0; i < arraySize; ++i) {
        a[i] = dist(gen);
        b[i] = dist(gen);
        control_sum1 += a[i];
        control_sum1 += b[i];
    }
    
    

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(a, b, out, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }



    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    int control_sum2=0;
    for (int i = 0; i < arraySize; i++) control_sum2 += out[i];
    std::cout << "suma kontrolna:\t" << control_sum1 << "\nsuma rzeczywista:\t"<<control_sum2;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *a, int *b, int *out, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_out = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

     cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    int block_size;
    int min_grid_size;
    int grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, addKernel, 0, size);

    grid_size = (size + block_size - 1) / block_size;

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<grid_size, block_size>>>(dev_a, dev_b, dev_out, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_out);
    return cudaStatus;
}
